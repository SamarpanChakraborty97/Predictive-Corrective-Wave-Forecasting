#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "math.h"
#include "SPH2DCPPCuda.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "stdio.h"

#define measureTransferTime (0) //use to time the tranfer and kernel execution


int moveDataToGPU(particleStructure* particles, paramsType* params, std::vector<kinematicsFunctionStructure>* kinematicsFunction, struct  particleStructure** ppdParticles, struct paramsType** ppdParams, struct kinematicsFunctionStructure** ppdKinematicsFunction, particleStructure** pdParticlesHostMirror) {


#if measureTransferTime
	hipEvent_t transferStart, transferStop;
	hipEventCreate(&transferStart);
	hipEventCreate(&transferStop);
#endif

#if measureTransferTime	
	hipEventRecord(transferStart, 0);
#endif


	//make an array of structs on the CPU
	//need to make on heap, since it is sized at runtime
	kinematicsFunctionStructure* H = (kinematicsFunctionStructure*)malloc((*params).nFunctions * sizeof(kinematicsFunctionStructure));

	//populate the fields of the array with the actual data
	for (int ind1 = 0; ind1 < (*params).nFunctions; ind1++)
	{
		H[ind1] = (*kinematicsFunction)[ind1];  //copy all fields
	};


	int nTotal = params->nTotal;
	int nFunctions = params->nFunctions;
	int nTime = params->nTime;
	int nFreqs = params->nFreqs;

	//allocate memory & copy params
	hipMalloc((void**)ppdParams, sizeof(paramsType));
	hipMemcpy(*ppdParams, params, sizeof(paramsType), hipMemcpyHostToDevice);

	//allocate memory & copy kinematics function structure
	hipMalloc((void**)ppdKinematicsFunction, (nFunctions) * sizeof(kinematicsFunctionStructure));
	hipMemcpy(*ppdKinematicsFunction, H, sizeof(kinematicsFunctionStructure) * nFunctions, hipMemcpyHostToDevice);
	free(H);  //we do not use the kinematics function for anything else


	//allocate memory on the GPU for all the fields of particles
	double* dx = 0; hipMalloc((void**)&dx, nTotal * sizeof(double));        hipMemcpy(dx, particles->x, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dy = 0; hipMalloc((void**)&dy, nTotal * sizeof(double));        hipMemcpy(dy, particles->y, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dvx = 0; hipMalloc((void**)&dvx, nTotal * sizeof(double));       hipMemcpy(dvx, particles->vx, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dvy = 0; hipMalloc((void**)&dvy, nTotal * sizeof(double));       hipMemcpy(dvy, particles->vy, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dfx = 0; hipMalloc((void**)&dfx, nTotal * sizeof(double));       hipMemcpy(dfx, particles->fx, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dfy = 0; hipMalloc((void**)&dfy, nTotal * sizeof(double));       hipMemcpy(dfy, particles->x, nTotal * sizeof(double), hipMemcpyHostToDevice);
	//double* dradius = 0; hipMalloc((void**)&dradius, nTotal * sizeof(double));   hipMemcpy(dradius, particles->radius, nTotal * sizeof(double), hipMemcpyHostToDevice);
	//double* dmass = 0; hipMalloc((void**)&dmass, nTotal * sizeof(double));     hipMemcpy(dmass, particles->mass, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* ddensity = 0; hipMalloc((void**)&ddensity, nTotal * sizeof(double));  hipMemcpy(ddensity, particles->density, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dpressure = 0; hipMalloc((void**)&dpressure, nTotal * sizeof(double));  hipMemcpy(dpressure, particles->pressure, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* docx = 0; hipMalloc((void**)&docx, nTotal * sizeof(double));      hipMemcpy(docx, particles->ocx, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* docy = 0; hipMalloc((void**)&docy, nTotal * sizeof(double));      hipMemcpy(docy, particles->ocy, nTotal * sizeof(double), hipMemcpyHostToDevice);
	//double* domx = 0; hipMalloc((void**)&domx, nTotal * sizeof(double));      hipMemcpy(domx, particles->omx, nTotal * sizeof(double), hipMemcpyHostToDevice);
	//double* domy = 0; hipMalloc((void**)&domy, nTotal * sizeof(double));      hipMemcpy(domy, particles->omy, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dpreviousX = 0; hipMalloc((void**)&dpreviousX, nTotal * sizeof(double)); hipMemcpy(dpreviousX, particles->previousX, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* dpreviousY = 0; hipMalloc((void**)&dpreviousY, nTotal * sizeof(double)); hipMemcpy(dpreviousY, particles->previousY, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* ddet_values = 0; hipMalloc((void**)&ddet_values, nTotal * sizeof(double)); hipMemcpy(ddet_values, particles->det_values, nTotal * sizeof(double), hipMemcpyHostToDevice);
	double* damps = 0; hipMalloc((void**)&damps, nFreqs * sizeof(double)); hipMemcpy(damps, particles->amps, nFreqs * sizeof(double), hipMemcpyHostToDevice);
	double* domegas = 0; hipMalloc((void**)&domegas, nFreqs * sizeof(double)); hipMemcpy(domegas, particles->omegas, nFreqs * sizeof(double), hipMemcpyHostToDevice);
	double* dtranFuncs = 0; hipMalloc((void**)&dtranFuncs, nFreqs * sizeof(double)); hipMemcpy(dtranFuncs, particles->tranFuncs, nFreqs * sizeof(double), hipMemcpyHostToDevice);
	double* dwavenumbers = 0; hipMalloc((void**)&dwavenumbers, nFreqs * sizeof(double)); hipMemcpy(dwavenumbers, particles->wavenumbers, nFreqs * sizeof(double), hipMemcpyHostToDevice);
	//double* dtimeSteps = 0; hipMalloc((void**)&dtimeSteps, nTime * sizeof(double)); hipMemcpy(dtimeSteps, particles->timeSteps, nTime * sizeof(double), hipMemcpyHostToDevice);
	//double* dL1 = 0; hipMalloc((void**)&dL1, nTotal * sizeof(double));          hipMemcpy(dL1, particles->L1, nTotal * sizeof(double), hipMemcpyHostToDevice);
	//double* drhoGradX = 0; hipMalloc((void**)&drhoGradX, nTotal * sizeof(double));          hipMemcpy(drhoGradX, particles->rhoGradX, nTotal * sizeof(double), hipMemcpyHostToDevice);

	int* dgridParticleHash = 0; hipMalloc((void**)&dgridParticleHash, nTotal * sizeof(int));  //there is nothing to copy
	int* dgridParticleIndex = 0; hipMalloc((void**)&dgridParticleIndex, nTotal * sizeof(int));
	int* cellStart = 0; hipMalloc((void**)&cellStart, params->nCellsTotal * sizeof(int));
	int* cellEnd = 0; hipMalloc((void**)&cellEnd, params->nCellsTotal * sizeof(int));
	//double* posDiv = 0; hipMalloc((void**)&posDiv, params->nTotal * sizeof(double));
	//double* shiftGradX = 0; hipMalloc((void**)&shiftGradX, params->nTotal * sizeof(double));
	//double* shiftGradY = 0; hipMalloc((void**)&shiftGradY, params->nTotal * sizeof(double));
	//double* sortedShift = 0; hipMalloc((void**)&sortedShift, params->nTotal * sizeof(double));
	//double* L1 = 0; hipMalloc((void**)&L1, params->nTotal * sizeof(double));
	//double* L2 = 0; hipMalloc((void**)&L2, params->nTotal * sizeof(double));
	//double* L3 = 0; hipMalloc((void**)&L3, params->nTotal * sizeof(double));
	//double* L4 = 0; hipMalloc((void**)&L4, params->nTotal * sizeof(double));
	//double* rhoGradX = 0; hipMalloc((void**)&rhoGradX, params->nTotal * sizeof(double));
	//double* rhoGradY = 0; hipMalloc((void**)&rhoGradY, params->nTotal * sizeof(double));
	double* sortedX = 0; hipMalloc((void**)&sortedX, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedY = 0; hipMalloc((void**)&sortedY, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedVx = 0; hipMalloc((void**)&sortedVx, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedVy = 0; hipMalloc((void**)&sortedVy, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedRho = 0; hipMalloc((void**)&sortedRho, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedA11 = 0; hipMalloc((void**)&sortedA11, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedA12 = 0; hipMalloc((void**)&sortedA12, nTotal * sizeof(double));  //there is nothing to copy
	double* sortedA22 = 0; hipMalloc((void**)&sortedA22, nTotal * sizeof(double));  //there is nothing to copy
	double* XSPHVelX = 0; hipMalloc((void**)&XSPHVelX, nTotal * sizeof(double));    //there is nothing to copy
	double* XSPHVelY = 0; hipMalloc((void**)&XSPHVelY, nTotal * sizeof(double));   //there is nothing to copy
	double* vxH = 0; hipMalloc((void**)&vxH, nTotal * sizeof(double));    //there is nothing to copy
	double* vyH = 0; hipMalloc((void**)&vyH, nTotal * sizeof(double));   //there is nothing to copy
	double* sortedPressure = 0; hipMalloc((void**)&sortedPressure, nTotal * sizeof(double));   //there is nothing to copy
	//double* pressure = 0; hipMalloc((void**)&pressure, nTotal * sizeof(double));   //there is nothing to copy
	double* sortedRhoFiltered = 0; hipMalloc((void**)&sortedRhoFiltered, nTotal * sizeof(double));   //there is nothing to copy
	double* sorteddRhodt = 0; hipMalloc((void**)&sorteddRhodt, nTotal * sizeof(double));   //there is nothing to copy
	

	//place the device pointers in the host mirror structure
	(*pdParticlesHostMirror)->x = dx;
	(*pdParticlesHostMirror)->y = dy;
	(*pdParticlesHostMirror)->vx = dvx;
	(*pdParticlesHostMirror)->vy = dvy;
	(*pdParticlesHostMirror)->fx = dfx;
	(*pdParticlesHostMirror)->fy = dfy;
	//(*pdParticlesHostMirror)->radius = dradius;
	//(*pdParticlesHostMirror)->mass = dmass;
	(*pdParticlesHostMirror)->density = ddensity;
	(*pdParticlesHostMirror)->pressure = dpressure;
	(*pdParticlesHostMirror)->ocx = docx;
	(*pdParticlesHostMirror)->ocy = docy;
	//(*pdParticlesHostMirror)->omx = domx;
	//(*pdParticlesHostMirror)->omy = domy;
	(*pdParticlesHostMirror)->previousX = dpreviousX;
	(*pdParticlesHostMirror)->previousY = dpreviousY;
	(*pdParticlesHostMirror)->det_values = ddet_values;
	(*pdParticlesHostMirror)->amps = damps;
	(*pdParticlesHostMirror)->omegas = domegas;
	(*pdParticlesHostMirror)->tranFuncs = dtranFuncs;
	(*pdParticlesHostMirror)->wavenumbers = dwavenumbers;
	//(*pdParticlesHostMirror)->timeSteps = dtimeSteps;
	//(*pdParticlesHostMirror)->L1 = dL1;
	//(*pdParticlesHostMirror)->rhoGradX = drhoGradX;	

	(*pdParticlesHostMirror)->gridParticleHash = dgridParticleHash;
	(*pdParticlesHostMirror)->gridParticleIndex = dgridParticleIndex;
	(*pdParticlesHostMirror)->cellStart = cellStart;
	(*pdParticlesHostMirror)->cellEnd = cellEnd;
	//(*pdParticlesHostMirror)->posDiv = posDiv;
	//(*pdParticlesHostMirror)->shiftGradX = shiftGradX;
	//(*pdParticlesHostMirror)->shiftGradY = shiftGradY;
	//(*pdParticlesHostMirror)->sortedShift = sortedShift;
	//(*pdParticlesHostMirror)->L1 = L1;
	//(*pdParticlesHostMirror)->L2 = L2;
	//(*pdParticlesHostMirror)->L3 = L3;
	//(*pdParticlesHostMirror)->L4 = L4;
	//(*pdParticlesHostMirror)->rhoGradX = rhoGradX;
	//(*pdParticlesHostMirror)->rhoGradY = rhoGradY;
	(*pdParticlesHostMirror)->sortedX = sortedX;
	(*pdParticlesHostMirror)->sortedY = sortedY;
	(*pdParticlesHostMirror)->sortedVx = sortedVx;
	(*pdParticlesHostMirror)->sortedVy = sortedVy;
	(*pdParticlesHostMirror)->sortedRho = sortedRho;
	(*pdParticlesHostMirror)->sortedA11 = sortedA11;
	(*pdParticlesHostMirror)->sortedA12 = sortedA12;
	(*pdParticlesHostMirror)->sortedA22 = sortedA22;
	(*pdParticlesHostMirror)->XSPHVelX = XSPHVelX;
	(*pdParticlesHostMirror)->XSPHVelY = XSPHVelY;
	(*pdParticlesHostMirror)->vxH = vxH;
	(*pdParticlesHostMirror)->vyH = vyH;
	(*pdParticlesHostMirror)->sortedPressure = sortedPressure;
	//(*pdParticlesHostMirror)->pressure = pressure;
	(*pdParticlesHostMirror)->sortedRhoFiltered = sortedRhoFiltered;
	(*pdParticlesHostMirror)->sorteddRhodt = sorteddRhodt;

	//allocate memory on the GPU for the particles structure
	hipMalloc((void**)ppdParticles, sizeof(particleStructure));

	//copy the device mirror structure from the host to the device
	hipMemcpy(*ppdParticles, *pdParticlesHostMirror, sizeof(particleStructure), hipMemcpyHostToDevice);



	/*



int numBytes         = sizeof(double)*nTotal;
double2* dPos        = 0;  //device X
double2* dPosCO      = 0;  //device X constrained original
double2* dSortedPos  = 0;
double2* dmassRadius = 0;  //device mass
double2* dVel        = 0;
double2* dVelHalf    = 0;
double2* dSortedVel  = 0;
double2* dForce      = 0;
double* dpRho        = 0;  //device rho
double* dsortedpRho  = 0;  //device rho
double2* dXSPHVel    = 0; //velocity diffusion from XSPH
int* dpColor         = 0;  //color is rearranged, this should be done on the GPU
int* dpColorSorted   = 0;  //color is rearranged, this should be done on the GPU
paramsType* dParams;
kinematicsFunctionStructure* dKinematicsFunction;


int* dGridParticleHash  = 0;
int* dGridParticleIndex = 0;
int* dCellStart         = 0;
int* dCellEnd           = 0;
int* dInd1              = 0;
int ind1 = 0;  //iteration governing the loop;  must be declared here to be sent to device

hipMalloc((void**)&dPos,nTotal*sizeof(double2));
hipMalloc((void**)&dPosCO,(*params).nConstrained*sizeof(double2));  //constrained original
hipMalloc((void**)&dSortedPos,nTotal*sizeof(double2));
hipMalloc((void**)&dmassRadius,nTotal*sizeof(double2));
hipMalloc((void**)&dVel,nTotal*sizeof(double2));
hipMalloc((void**)&dVelHalf,nTotal*sizeof(double2));
hipMalloc((void**)&dSortedVel,nTotal*sizeof(double2));
hipMalloc((void**)&dXSPHVel,nTotal*sizeof(double2));

hipMalloc((void**)&dForce,nTotal*sizeof(double2));
hipMalloc((void**)&dpRho,numBytes);
hipMalloc((void**)&dsortedpRho,numBytes);
hipMalloc((void**)&dpColor,nTotal*sizeof(int));
hipMalloc((void**)&dpColorSorted,nTotal*sizeof(int));
hipMalloc((void**)&dGridParticleHash,nTotal*sizeof(int));
hipMalloc((void**)&dGridParticleIndex,nTotal*sizeof(int));
hipMalloc((void**)&dCellStart,(*params).nCellsTotal*sizeof(int));
hipMalloc((void**)&dCellEnd,(*params).nCellsTotal*sizeof(int));
hipMalloc((void**)&dInd1,sizeof(int));
hipMalloc((void**)&dParams,sizeof(paramsType));
hipMalloc((void**)&dKinematicsFunction,(*params).nFunctions*sizeof(kinematicsFunctionStructure));

hipMemcpy(dPos,pos,nTotal*sizeof(double2),hipMemcpyHostToDevice);
hipMemcpy(dPosCO,posCO,(*params).nConstrained*sizeof(double2),hipMemcpyHostToDevice);
hipMemcpy(dmassRadius,massRadius,nTotal*sizeof(double2),hipMemcpyHostToDevice);
hipMemcpy(dpRho,pRho,numBytes,hipMemcpyHostToDevice);
hipMemcpy(dInd1,&ind1,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dVel,vel,nTotal*sizeof(double2),hipMemcpyHostToDevice);  //just copying zeros
hipMemcpy(dForce,force,nTotal*sizeof(double2),hipMemcpyHostToDevice);  //just copying zeros
hipMemcpy(dpColor,pColor,nTotal*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dParams,params,sizeof(paramsType),hipMemcpyHostToDevice);
hipMemcpy(dKinematicsFunction,H,(*params).nFunctions*sizeof(kinematicsFunctionStructure),hipMemcpyHostToDevice);


*/

	return 0;
}